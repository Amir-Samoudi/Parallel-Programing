#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;
using namespace cv;

void applyEmbossFilterSerial(const Mat& input, Mat& output) {
    int emboss_kernel[3][3] = {
        {-2, -1, 0},
        {-1,  1, 1},
        { 0,  1, 2}
    };

    int kernel_size = 3;

    for (int y = 0; y < input.rows - kernel_size - 1; y++) {
        for (int x = 0; x < input.cols - kernel_size - 1; x++) {
            for (int c = 0; c < input.channels(); c++) {
                int sum = 0;

                for (int i = 0; i < kernel_size; i++) {
                    for (int j = 0; j < kernel_size; j++) {
                        int pixel_value = input.at<Vec3b>(y + i, x + j)[c];
                        sum += pixel_value * emboss_kernel[i][j];
                    }
                }

                output.at<Vec3b>((y + 1), (x + 1))[c] = saturate_cast<uchar>(sum);
            }
        }
    }
}


__global__ void applyEmbossFilterCUDAKernel(unsigned char* input, unsigned char* output, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    //if (x < 1 || x >= width - 1 || y < 1 || y >= height - 1) return;

    //int emboss_kernel[3][3] = {
    //    {-2, -1, 0},
    //    {-1,  1, 1},
    //    { 0,  1, 2}
    //};

    for (int c = 0; c < channels; c++) {
        //int sum = 0;

        //for (int i = -1; i <= 1; i++) {
        //    for (int j = -1; j <= 1; j++) {
        //        int pixel_value = input[((y + i) * width + (x + j)) * channels + c];
        //        sum += pixel_value * emboss_kernel[i + 1][j + 1];
        //    }
        //}

        output[(y * width + x) * channels + c] = min(max(sum, 0), 255);
        output[(y * width + x) * channels + c] = min(max(sum, 0), 255);
    }
}


void applyEmbossFilterCUDA(const Mat& input, Mat& output) {
    int width = input.cols;
    int height = input.rows;
    int channels = input.channels();

    size_t size = width * height * channels * sizeof(unsigned char);

   
    unsigned char* d_input, * d_output;
    hipMalloc((void**)&d_input, size);
    hipMalloc((void**)&d_output, size);

    hipMemcpy(d_input, input.data, size, hipMemcpyHostToDevice);


    dim3 block_size(16, 16);
    dim3 grid_size((width + block_size.x - 1) / block_size.x, (height + block_size.y - 1) / block_size.y);


    applyEmbossFilterCUDAKernel <<<grid_size, block_size >>> (d_input, d_output, width, height, channels);


    hipDeviceSynchronize();

    hipMemcpy(output.data, d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

int main() {
   
    string image_path = "image.jpg";
    Mat image = imread(image_path, IMREAD_COLOR);

    if (image.empty()) {
        cerr << "Error: Unable to load image!" << endl;
        return -1;
    }

    Mat output_serial = image.clone();
    Mat output_cuda = image.clone();

    
    auto start_serial = chrono::high_resolution_clock::now();
    applyEmbossFilterSerial(image, output_serial);
    auto end_serial = chrono::high_resolution_clock::now();
    chrono::duration<double, milli> elapsed_serial = end_serial - start_serial;

   
    hipEvent_t start_cuda, stop_cuda;
    hipEventCreate(&start_cuda);
    hipEventCreate(&stop_cuda);

    hipEventRecord(start_cuda);
    applyEmbossFilterCUDA(image, output_cuda);
    hipEventRecord(stop_cuda);

    hipEventSynchronize(stop_cuda);
    float elapsed_cuda;
    hipEventElapsedTime(&elapsed_cuda, start_cuda, stop_cuda);

    
    imwrite("emboss_effect_serial.jpg", output_serial);
    imwrite("emboss_effect_cuda.jpg", output_cuda);

    cout << "Serial Execution Time: " << elapsed_serial.count() << " ms" << endl;
    cout << "CUDA Execution Time: " << elapsed_cuda << " ms" << endl;
    cout << "Speedup: " << elapsed_serial.count() / elapsed_cuda << "x" << endl;

    return 0;
}
